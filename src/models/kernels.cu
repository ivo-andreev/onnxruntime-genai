// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits>

namespace Generators {
namespace cuda {

template <typename T>
__global__ void UpdatePositionIds(T* positions, int batch_beam_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < batch_beam_size)
    positions[i]++;
}

template <typename T>
void Launch_UpdatePositionIds(T* positions, int batch_beam_size, hipStream_t stream) {
  UpdatePositionIds<T><<<(batch_beam_size + 255) / 256, 256, 0, stream>>>(positions, batch_beam_size);
}

template void Launch_UpdatePositionIds(int32_t* positions, int batch_beam_size, hipStream_t stream);
template void Launch_UpdatePositionIds(int64_t* positions, int batch_beam_size, hipStream_t stream);

template <typename T>
__global__ void CopyAndUpdateAttentionMask(T* mask_data, const T* old_mask_data, int batch_beam_size,
                                           int current_length, int max_length) {
  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  int i = global_index / current_length;
  int j = global_index % current_length;
  if (i < batch_beam_size) {
    if (j < current_length - 1) {
      mask_data[i * max_length + j] = old_mask_data[i * (current_length - 1) + j];
    } else {
      mask_data[i * max_length + j] = 1;
    }
  }
}

template <typename T>
__global__ void UpdateAttentionMask(T* mask_data, int batch_beam_size, int current_length, int max_length) {
  int i = blockIdx.x;
  if (i < batch_beam_size) {
    mask_data[i * max_length + current_length] = 1;
  }
}

template <typename T>
void Launch_UpdateAttentionMask(T* mask_data, const T* old_mask_data, int batch_beam_size, int current_length,
                                int max_length, bool update_only, hipStream_t stream) {
  if (update_only) {
    UpdateAttentionMask<T>
        <<<batch_beam_size, 1, 0, stream>>>(mask_data, batch_beam_size, current_length, max_length);
  } else {
    CopyAndUpdateAttentionMask<T><<<(batch_beam_size * max_length + 255) / 256, 256, 0, stream>>>(
        mask_data, old_mask_data, batch_beam_size, current_length, max_length);
  }
}

template void Launch_UpdateAttentionMask(int32_t* mask_data, const int32_t* old_mask_data, int batch_beam_size,
                                         int current_length, int max_length, bool update_only, hipStream_t stream);
template void Launch_UpdateAttentionMask(int64_t* mask_data, const int64_t* old_mask_data, int batch_beam_size,
                                         int current_length, int max_length, bool update_only, hipStream_t stream);

// Support head_size up to 128
constexpr unsigned int kTileSize = 32;
constexpr unsigned int kSeqTileSize = 16;

__global__ void ReorderPastStatesKernel(float4* out_buffer,
                                        const float4* in_buffer,
                                        int batch_size,
                                        int num_heads,
                                        int max_length,
                                        int chunked_head_size) {
  __shared__ float4 tile[kSeqTileSize][kTileSize + 1];

  const int b = blockIdx.z;
  const int n = blockIdx.y;
  const int s_base = blockIdx.x * kSeqTileSize;
  const int s = s_base + threadIdx.y;
  const int base_offset = (b * num_heads + n) * max_length * chunked_head_size;

  if (s < max_length) {
    const int in_offset = base_offset + s * chunked_head_size + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = in_buffer[in_offset];
  }

  __syncthreads();

  const int tidx = threadIdx.x + threadIdx.y * chunked_head_size;
  const int tidx_x = tidx % kSeqTileSize;
  const int tidx_y = tidx / kSeqTileSize;

  const int s2 = s_base + tidx_x;

  if (s2 < max_length) {
    const int out_offset = base_offset + tidx_y * max_length + s2;
    out_buffer[out_offset] = tile[tidx_x][tidx_y];
  }
}

void ReorderPastStatesKernelLauncher(void* out_buffer,
                                     const void* in_buffer,
                                     int batch_size,
                                     int num_heads,
                                     int max_length,
                                     int head_size,
                                     int chunk_size,
                                     hipStream_t stream) {
  // [B, N, max_length, H2(head_size/chunk_size), equv_chunk_size] -> [B, N, H2(head_size/chunk_size), max_length, equv_chunk_size]
  const int chunked_head_size = head_size / chunk_size;
  const dim3 block(chunked_head_size, kSeqTileSize);
  const dim3 grid((max_length + kSeqTileSize - 1) / kSeqTileSize, num_heads, batch_size);
  if (chunk_size == 4 || chunk_size == 8) {
    ReorderPastStatesKernel<<<grid, block, 0, stream>>>(reinterpret_cast<float4*>(out_buffer),
                                                        reinterpret_cast<const float4*>(in_buffer),
                                                        batch_size,
                                                        num_heads,
                                                        max_length,
                                                        chunked_head_size);
  }
}

__global__ void HandleEOSArray(float* batch_logits, int batch_beam_size, int vocab_size, const int32_t* eos_token_ids, int eos_token_ids_count) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= batch_beam_size)
    return;

  float* logits = batch_logits + index * vocab_size;
  float max = std::numeric_limits<float>::lowest();
  for (int i = 0; i < eos_token_ids_count; i++) {
    max = std::max(max, logits[eos_token_ids[i]]);
    logits[eos_token_ids[i]] = std::numeric_limits<float>::lowest();  // Set all EOS token options to never happen (the first will get the max of all)
  }

  logits[eos_token_ids[0]] = max;  // Set the score of the primary EOS token to the highest of any of the EOS tokens
}

void LaunchHandleEOSArray(float* batch_logits, int batch_beam_size, int vocab_size, const int32_t* eos_token_ids, int eos_token_ids_count, hipStream_t stream) {
  HandleEOSArray<<<(batch_beam_size + 255) / 256, 256, 0, stream>>>(batch_logits, batch_beam_size, vocab_size, eos_token_ids, eos_token_ids_count);
}

__global__ void ConvertFp16ToFp32(const half* src, float* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    dst[idx] = __half2float(src[idx]);
}

void LaunchFp16ToFp32(const uint16_t* fp16, float* fp32, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertFp16ToFp32<<<num_blocks, block_size, 0, stream>>>(reinterpret_cast<const half*>(fp16), fp32, count);
}

__global__ void ConvertFp32ToFp16(const float* src, half* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    dst[idx] = __float2half(src[idx]);
}

void LaunchFp32ToFp16(const float* fp32, uint16_t* fp16, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertFp32ToFp16<<<num_blocks, block_size, 0, stream>>>(fp32, reinterpret_cast<half*>(fp16), count);
}

__global__ void ConvertInt32ToInt64(const int32_t* src, int64_t* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count) {
    dst[idx] = src[idx];
  }
}

void LaunchInt32ToInt64(const int32_t* src, int64_t* dst, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertInt32ToInt64<<<num_blocks, block_size, 0, stream>>>(src, dst, count);
}

__global__ void UpdateDecoderMaskedMultiHeadAttentionCacheIndirectionKernel(int32_t* tgt_indir_cache,
                                                                            const int32_t* src_indir_cache,
                                                                            const int32_t* beam_ids,
                                                                            int batch_size,
                                                                            int beam_width,
                                                                            int input_seq_length,
                                                                            int max_seq_length,
                                                                            int current_length) {
  int time_step = threadIdx.x + blockIdx.x * blockDim.x;
  int bb_id = threadIdx.y + blockIdx.y * blockDim.y;
  const int batch_id = bb_id / beam_width;
  const int beam_id = bb_id % beam_width;

  if (bb_id >= beam_width * batch_size || time_step >= current_length) {
    return;
  }

  const int src_beam = beam_ids[batch_id * beam_width + beam_id] % beam_width;

  const int tgt_offset = batch_id * beam_width * max_seq_length + beam_id * max_seq_length + time_step;

  if (time_step < input_seq_length) {
    // For time steps that correspond to the input sequence,
    // the beam that it comes from is always 0.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(0);
  } else if (time_step == (current_length - 1)) {
    // For the final (newly generated) time step,
    // the beam that it comes from is always the beam that we
    // are currently processing (i.e.) from this point on, these time-steps
    // form the new beams.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(beam_id);
  } else {
    // For all other time-steps, we look up the source indirection, to
    // see which beam it came from based on the `src_beam`.
    const int src_offset = batch_id * beam_width * max_seq_length + src_beam * max_seq_length + time_step;
    tgt_indir_cache[tgt_offset] = src_indir_cache[src_offset];
  }
}

void UpdateDecoderMaskedMultiHeadAttentionCacheIndirection(int32_t* tgt_indir_cache,
                                                           const int32_t* src_indir_cache,
                                                           const int32_t* beam_ids,
                                                           int batch_size,
                                                           int beam_width,
                                                           int input_seq_length,
                                                           int max_seq_length,
                                                           int current_length,
                                                           hipStream_t stream) {
  const dim3 block(32);
  const dim3 grid((current_length + block.x - 1) / block.x, batch_size * beam_width);
  UpdateDecoderMaskedMultiHeadAttentionCacheIndirectionKernel<<<grid, block, 0, stream>>>(tgt_indir_cache,
                                                                                          src_indir_cache,
                                                                                          beam_ids,
                                                                                          batch_size,
                                                                                          beam_width,
                                                                                          input_seq_length,
                                                                                          max_seq_length,
                                                                                          current_length);
}

}  // namespace cuda
}  // namespace Generators
